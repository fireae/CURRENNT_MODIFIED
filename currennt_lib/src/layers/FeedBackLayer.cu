#include "hip/hip_runtime.h"
/******************************************************************************
 * This file is an addtional component of CURRENNT. 
 * Xin WANG
 * National Institute of Informatics, Japan
 * 2016
 *
 * This file is part of CURRENNT. 
 * Copyright (c) 2013 Johannes Bergmann, Felix Weninger, Bjoern Schuller
 * Institute for Human-Machine Communication
 * Technische Universitaet Muenchen (TUM)
 * D-80290 Munich, Germany
 *
 *
 * CURRENNT is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * CURRENNT is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with CURRENNT.  If not, see <http://www.gnu.org/licenses/>.
 *****************************************************************************/

#include "FeedBackLayer.hpp"

#include "../helpers/getRawPointer.cuh"
#include "../helpers/Matrix.hpp"
#include "../helpers/JsonClasses.hpp"
#include "../activation_functions/Logistic.cuh"
#include "../activation_functions/Tanh.cuh"

#include <thrust/transform.h>
#include <thrust/transform_reduce.h>
#include <thrust/for_each.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/fill.h>
#include <boost/foreach.hpp>
#include <boost/shared_ptr.hpp>
#include <boost/algorithm/string.hpp>
#include <boost/lexical_cast.hpp>
#include <vector>
#include <stdexcept>

#include "../Configuration.hpp"

#define FEEDBACKLAYER_DEBUG 0

namespace internal{
namespace {

    typedef activation_functions::Tanh     cell_act_fn_t;
    
    // dustbin.txt/Block1226x02
    
    struct vectorFillForward
    {
	// Copy the output of preceding layer to the output of this layer
	// Copy the output of target layer to the output of this layer

	int dimInput1;      // dimension of output of preceding layer
	int dimInput2;      // dimension of output of target layer (to be fed back, in total dim)
	int dimInput2Start; // from which dimension of the target to load (may not be 0)
	
	int dimOutput;      // dimension of output of this layer
	int parallel;       // number of parallel sentences

	int dim1Step;
	
	real_t *input1;     // preceding layer
	real_t *input2;     // target layer
	real_t *output;     // this layer

	int    *lookBack;   // lookback step
	int     lookBackStepNM; // how many steps to look back ?
	int     crossBoundary;
	// dispatched over Dim * T * Parallel
	__host__ __device__ void operator() (const thrust::tuple<const real_t&, int> &t)
	{
	    int outputEffIdx = t.get<1>();
	    int timeStep     = outputEffIdx / dimOutput;
	    int dimIdx       = outputEffIdx % dimOutput;

	    // Idx in the output of this layer
	    int outputIdx    = timeStep * dimOutput + dimIdx;
	    int lookBackTime = 0;

	    if (dimIdx < (dimInput1 + lookBackStepNM * dim1Step)){
		if (dimIdx >= dimInput1){
		    // copy from the target layer (feedback part)
		    
		    // get the dimension index (across multiple time steps)
		    dimIdx       = dimIdx - dimInput1;
		    
		    // get the time shift to be looked backwards
		    if (lookBack != NULL)
			lookBackTime = lookBack[dimIdx / dim1Step] * parallel;
		    else
			lookBackTime = 1;
		    
		    // get the dimension index in each time step
		    dimIdx       = dimIdx % dim1Step;
		    
		    if (timeStep < lookBackTime)      // loopback one step
			output[outputIdx] = 0.0;
		    else{
			output[outputIdx] = input2[(timeStep - lookBackTime) * dimInput2 +
						   dimIdx + dimInput2Start];
			
			// crossBoundary should be deleted
			if (crossBoundary == 3 &&
			    input2[(timeStep - lookBackTime) * dimInput2 + dimInput2Start] > 0.98){
			    output[outputIdx] = 0.0;
			    // Set the feedback to zero if previous frame is silence
			}
			//
		    }
		    
		}else{
		    //output[outputIdx] = 0;
		    output[outputIdx] = input1[timeStep * dimInput1 + dimIdx];
		}
	    }else{
		// this is section for aggregating information
	    }
	}
    };
    

    struct vectorAggregateForward
    {
	// Copy the output of preceding layer to the output of this layer
	// Copy the output of target layer to the output of this layer

	int dimInput2;      // dimension of output of target layer (to be fed back, in total dim)
	int dimInput2Start; // from which dimension of the target to load (may not be 0)
	
	int dimOutput;      // dimension of output of this layer
	int dimOutputStart;
	int dim1Band;
	int bandNum;
	
	real_t *input2;     // target layer
	real_t *output;     // this layer

	char   *boundaryInfo;
	int     startTime;
	int     endTime;

	real_t *aggBuffer;
	
	int   crossBoundary; // deliver the aggregation across the boundary

	// dispatched over Dim * Band
	__host__ __device__ void operator() (const thrust::tuple<const real_t&, int> &t)
	{
	    int     dimIdxRel  = t.get<1>();                             // relative dimension
	    
	    int     outputIdx  = dimOutputStart + dimIdxRel;           
	    int     inputIdx   = dimInput2Start + dimIdxRel % dim1Band;
	    
	    int     bandIdx    = dimIdxRel / dim1Band;           // which band this dimension is in?
	    int     boundTime  = 0;
	    real_t  aggreInfo  = 0.0;
	    
	    
	    for (int time = startTime ; time < endTime; time++){
		
		if (crossBoundary){
		    // deliver the aggregation across boundary
		    output[outputIdx]= aggreInfo;
		    if (boundaryInfo[time * bandNum + bandIdx] < 1 || time < 1){
			aggreInfo    = 0.0;  
			boundTime    = time; 
		    }
		}else{
		    // not deliver across boundary
		    if (boundaryInfo[time * bandNum + bandIdx] < 1 || time < 1){
			aggreInfo    = 0.0;  
			boundTime    = time; 
		    }
		    output[outputIdx]= aggreInfo;
		}

		if (crossBoundary == 3 && (inputIdx - dimInput2)>0 &&
		    input2[inputIdx - dimInput2 - dimIdxRel % dim1Band] > 0.98){
		    output[outputIdx] = 0;
		    // set the previous frame to zero if it is silence
		}
		
		if (crossBoundary == 3 && input2[inputIdx - dimIdxRel % dim1Band] > 0.98){
		    // don't aggregate this frame
		}else{
		    // aggregating information using tanh and moving average
		    aggreInfo  = (((time - boundTime) / ((time - boundTime)+1.0)) * aggreInfo +
				  cell_act_fn_t::fn(input2[inputIdx]) / ((time-boundTime)+1.0));
		}
		outputIdx += dimOutput;
		inputIdx  += dimInput2;
	    }
	}
    };


    struct vectorAggregateForwardInfer
    {
	int dimInput2;      // dimension of output of target layer (to be fed back, in total dim)
	int dimInput2Start; // from which dimension of the target to load (may not be 0)
	
	int dimOutput;      // dimension of output of this layer
	int dimOutputStart;
	int dim1Band;
	int bandNum;
	
	real_t *input2;     // target layer
	real_t *output;     // this layer

	char   *boundaryInfo;
	int     startTime;
	int     endTime;

	real_t *aggBuffer;
	
	int   crossBoundary; // deliver the aggregation across the boundary

	// dispatched over Dim * Band
	__host__ __device__ void operator() (const thrust::tuple<const real_t&, int> &t)
	{
	    int     dimIdxRel  = t.get<1>();                             // relative dimension
	    
	    /*********************** FATAL ERROR ***********************************
	       Fatal Error: for inferrence stage, outputIdx and inputIdx should point
	       to current time step:
	       for (int time = 0; time < startTime; time++)
	           outputIdx += dimOutput
                   inputIdx  += dimInput2
	    ************************************************************************/
	    // Error: 
	    //int     outputIdx  = dimOutputStart + dimIdxRel;           
	    //int     inputIdx   = dimInput2Start + dimIdxRel % dim1Band;
	    // Modified
	    int     outputIdx = startTime * dimOutput + dimOutputStart + dimIdxRel;
	    int     inputIdx  = startTime * dimInput2 + dimInput2Start + dimIdxRel % dim1Band; 
	    
	    int     bandIdx   = dimIdxRel / dim1Band;      // which band this dimension is in?	    
	    int     preTime   = 0;

	    // after the first frame
	    // take the accumulation the frames before the previous frame
	    real_t  aggreInfo = aggBuffer[dimIdxRel];      
	    int     boundTime = aggBuffer[bandNum * dim1Band + dimIdxRel];
	    
	    
	    for (int time = startTime ; time < endTime; time++){
		
		preTime = time - 1;
		if (preTime < 0){
		    // the first frame
		    output[outputIdx]     = 0.0;
		    aggBuffer[dimIdxRel]  = 0.0;
		    aggBuffer[bandNum * dim1Band + dimIdxRel] = time;
		    
		}else{
		    
		    // aggregating the previous frame
		    if (crossBoundary == 3 &&
			input2[inputIdx - dimInput2 - dimIdxRel % dim1Band] > 0.98){
			
			
		    }else{
			aggreInfo  = ((preTime-boundTime) / (preTime - boundTime + 1.0)) *
			    aggreInfo +
			    cell_act_fn_t::fn(input2[inputIdx - dimInput2]) /
			    (preTime - boundTime+1.0);
		    }
		    
		    // propagate the info to the current frame
		    if (crossBoundary == 3 &&
			input2[inputIdx - dimInput2 - dimIdxRel % dim1Band] > 0.98){
			output[outputIdx]= 0;
			
		    }else if (crossBoundary == 1){
			// deliver the aggregation across boundary
			output[outputIdx]= aggreInfo;
			if (boundaryInfo[time * bandNum + bandIdx] < 1){
			    aggreInfo    = 0.0;  
			    boundTime    = time; 
			}
		    }else{
			// not deliver across boundary
			if (boundaryInfo[time * bandNum + bandIdx] < 1){
			    aggreInfo    = 0.0;  
			    boundTime    = time; 
			}
			output[outputIdx]= aggreInfo;
		    }
		    // save the aggregation information for next time (during generation)
		    aggBuffer[dimIdxRel] = aggreInfo;
		    aggBuffer[bandNum * dim1Band + dimIdxRel] = boundTime; 
		}
		outputIdx += dimOutput;
		inputIdx  += dimInput2;
	    }
	}
    };

    
    struct vectorFillBackward
    {
	int dimInput1;      // dimension of the preceding layer
	int dimOutput;      // dimension of this layer
	
	real_t *outputError;
	
	// dispatched over Dim * T * Parallel
	// Dim here is the dimension of the previous layer
	__host__ __device__ real_t operator() (const int &outputIdx) const
	{
	    int timeStep  = outputIdx / dimInput1;
	    int dimIdx    = outputIdx % dimInput1;
	    return outputError[timeStep * dimOutput + dimIdx];
	}
    };
    
}
}

namespace layers{

    // dustbin.txt/Block 1226x01
    int ParseLayerOpt(const std::string options){
	std::vector<std::string> tempArgs;
	boost::split(tempArgs, options, boost::is_any_of("_"));
	return boost::lexical_cast<int>(tempArgs[0]);
    }

    void ParseLookBackStep(const std::string options, Cpu::int_vector &optVec){
	std::vector<std::string> tempArgs;
	boost::split(tempArgs, options, boost::is_any_of("_"));
	optVec.resize(tempArgs.size(), 0);
	for (int i =0 ; i<tempArgs.size(); i++)
	    optVec[i] = boost::lexical_cast<int>(tempArgs[i]);
    }

    void ConvertBoundaryInfo(Cpu::pattype_vector &boundary, Cpu::pattype_vector &distance,
			     Cpu::int_vector & aggOpt, const int curMaxLength)
    {
	// The boundary information logs the distance of this frame to the previous boundary
	// ex. 0 1 2 3 4 .. 10 0 1 2 .. 32 0
	std::vector<int> outTemp(aggOpt.size(), 0);
	for (int time = 0; time < curMaxLength; time++){
	    for (int band = 0; band < aggOpt.size(); band++){
		if (boundary[time] & (0b01 << aggOpt[band]))
		    outTemp[band] = 0;
		else
		    outTemp[band] = outTemp[band] + 1;
		distance[time * aggOpt.size() + band] = outTemp[band];
	    }
	}
    }
    
    template <typename TDevice>
    FeedBackLayer<TDevice>::FeedBackLayer(const helpers::JsonValue &layerChild,
					  const helpers::JsonValue &weightsSection,
					  Layer<TDevice>           &precedingLayer
					  )
	: TrainableLayer<TDevice>(layerChild, weightsSection, 0, 0, precedingLayer)
	, m_targetDim   (-1)
	, m_targetLayer (NULL)
    {
	m_targetBuffer.clear();
	
	const Configuration &config = Configuration::instance();
	
	// get ClockRNN state
	m_lookBackStr = ((layerChild->HasMember("lookback")) ? 
			 ((*layerChild)["lookback"].GetString()) : (""));
	if (m_lookBackStr.size()){
	    if (m_lookBackStr.size()==1 && m_lookBackStr[0] == '0'){
		// special case where lookback is not used
		m_lookBack.clear();
	    }else{
		// when lookback is explicitly specified
		cpu_int_vector tempOpt;
		ParseLookBackStep(m_lookBackStr, tempOpt);
		m_lookBack = tempOpt;
	    }
	}else{
	    // default only look back 1 step
	    m_lookBack.resize(1,1); 
	}

	// get aggregation information
	m_aggStr         = ((layerChild->HasMember("aggregate")) ? 
			    ((*layerChild)["aggregate"].GetString()) : (""));
	m_crossBoundary  = (layerChild->HasMember("aggregate_cross_boundary") ? 
			    (*layerChild)["aggregate_cross_boundary"].GetInt() : 0);

	if (m_aggStr.size()){
	    // configuratio for F0 aggregation
	    cpu_int_vector tempOpt;
	    ParseLookBackStep(m_aggStr, tempOpt);
	    m_aggOpt = tempOpt;
	    m_boundaryInfo.resize(m_aggOpt.size() * precedingLayer.maxSeqLength(), 0);
	    m_aggOptSyn      = config.aggregateOpt();
	}else{
	    // default, don't use aggregate
	    m_aggOpt.clear(); 
	}
    }

    template <typename TDevice>
    FeedBackLayer<TDevice>::~FeedBackLayer()
    {
    }

    template <typename TDevice>
    void FeedBackLayer<TDevice>::exportLayer(const helpers::JsonValue     &layersArray, 
					     const helpers::JsonAllocator &allocator) const
    {
        TrainableLayer<TDevice>::exportLayer(layersArray, allocator);
        (*layersArray)[layersArray->Size() - 1].AddMember("lookback",  m_lookBackStr.c_str(),
							  allocator);
        (*layersArray)[layersArray->Size() - 1].AddMember("aggregate", m_aggStr.c_str(),
							  allocator);
        (*layersArray)[layersArray->Size() - 1].AddMember("aggregate_cross_boundary", 
							  m_crossBoundary,
							  allocator);
    }

    template <typename TDevice>
    void FeedBackLayer<TDevice>::linkTargetLayer(Layer<TDevice> &targetLayer)
    {
	m_targetDim      = ParseLayerOpt(targetLayer.layerAddInfor(1));
	m_targetLayer    = &targetLayer;

	// Now, use all target features for feedback
	// To be completed
	m_targetDimStart = 0;
	m_targetDimEnd   = m_targetDim;

	// dim * look_back + dim * aggregate + preceding_layer
	int dimExpected = ((m_targetDimEnd - m_targetDimStart) * m_lookBack.size() +
			   (m_targetDimEnd - m_targetDimStart) * m_aggOpt.size()   +
			   this->precedingLayer().size());
	
	if (dimExpected !=this->size()){
	    printf("Feedback dim + Feedforward dim = %d\n", dimExpected);
	    throw std::runtime_error("Error in network.jsn feedback layer size");
	}
	if (m_targetDimEnd > m_targetDim || m_targetDimStart > m_targetDim ||
	    m_targetDimEnd < m_targetDimStart){
	    throw std::runtime_error("Error in configuration of targetDimStart, targetDimEnd");
	}

	// initialize m_aggBuffer
	//     m_aggBuffer stores the intermediate state of aggregation and the previous boundary
	//     time
	if (m_aggOpt.size())
	    m_aggBuffer.resize((m_targetDimEnd - m_targetDimStart) * m_aggOpt.size() * 2, 0.0);
	
	// print information
	printf("\nCreating the feedback link:\n");
	printf("\tFrom %s [%d-%d]", targetLayer.type().c_str(), m_targetDimStart, m_targetDimEnd);
	printf("\tLook Back [%s]", m_lookBackStr.c_str());
	if (m_aggOpt.size()){
	    printf("\tAggregating [%s]", m_aggStr.c_str());
	    if (m_crossBoundary)
		printf(" cross boundary");
	}
	printf("\n");
    }

    template <typename TDevice>
    void FeedBackLayer<TDevice>::loadSequences(const data_sets::DataSetFraction &fraction)
    {
	TrainableLayer<TDevice>::loadSequences(fraction);

	// read in the boundary information
	if (m_aggStr.size()){
	    //
	    if (this->parallelSequences()>1){
		printf("Please use parallel_sequences = 1\n");
		throw std::runtime_error("Not implemented: F0 aggregation for parallel training");
	    }
	    
	    if (fraction.auxDataDim()>0){
		if (m_aggOpt.size() > CHAR_BIT)
		    throw std::runtime_error("Aggregate information is larger than CHAR_BIT");

		// Read in the aux label information
		Cpu::pattype_vector auxInfo = fraction.auxPattypeData();
		if (auxInfo.size() != this->curMaxSeqLength())
		    throw std::runtime_error("Error unequal length of clockTime size");
	    
		// Convert the boundary information into distance information
		Cpu::pattype_vector tempDistance(m_boundaryInfo.size(), 0);
		cpu_int_vector      tmpAggOpt = m_aggOpt;
		ConvertBoundaryInfo(auxInfo, tempDistance, tmpAggOpt, this->curMaxSeqLength());
		m_boundaryInfo = tempDistance;
		
		if (FEEDBACKLAYER_DEBUG){
		    for (int i = 0; i < this->curMaxSeqLength(); i++){
			printf("%d:%3d\t", i, auxInfo[i]);
			for (int j = 0; j<m_aggOpt.size(); j++)
			    printf("%3d ", tempDistance[i*m_aggOpt.size()+j]);
			printf("\n");
		    }
		}
		
		// prepare the aggregate buffer (which will be used in generation)
		m_aggBuffer.resize((m_targetDimEnd - m_targetDimStart) * m_aggOpt.size() * 2, 0.0);
	    }else {
		throw std::runtime_error("No boundary information is provided");
	    }
	}else{
	    // nothing if aggregation is not used
	}
    }
    
    template <typename TDevice>
    const std::string& FeedBackLayer<TDevice>::type() const
    {
        static std::string s;
        if (s.empty()) s = "feedback";
        return s;
    }

    // computeForward: 
    //  in training stage, target data are known
    template <typename TDevice>
    void FeedBackLayer<TDevice>::computeForwardPass()
    {
	if (m_targetLayer == NULL)
	    throw std::runtime_error("Target layer is not linked");
	
	thrust::fill(this->outputs().begin(), this->outputs().end(), 0.0);
	{{
	    // Concatenate the output of the preceding layer and the feedback layer
	    int previousSize  = this->precedingLayer().size();
	    
	    internal::vectorFillForward fn;
	    fn.dimInput1      = previousSize;     // the dimension from preceding layer
	    
	    fn.dimInput2      = m_targetDim;      // the dimension of the output of target layer
	    fn.dimInput2Start = m_targetDimStart; // from which dimension to load from target layer
	    fn.dim1Step       = m_targetDimEnd - m_targetDimStart; // dimension for 1 step
		
	    fn.dimOutput      = this->size();     
	    fn.parallel       = this->parallelSequences();

	    fn.input1         = helpers::getRawPointer(this->precedingLayer().outputs());
	    fn.input2         = helpers::getRawPointer(m_targetLayer->secondOutputs(true));
	    fn.output         = helpers::getRawPointer(this->outputs());
	    fn.lookBack       = helpers::getRawPointer(this->m_lookBack);

	    fn.lookBackStepNM = this->m_lookBack.size();
	    fn.crossBoundary  = m_crossBoundary;
	    int n = this->curMaxSeqLength() * this->parallelSequences() * this->size();
	    thrust::for_each(
		thrust::make_zip_iterator(thrust::make_tuple(this->outputs().begin(),
							     thrust::counting_iterator<int>(0))),
		thrust::make_zip_iterator(thrust::make_tuple(this->outputs().begin()+n,
							     thrust::counting_iterator<int>(0)+n)),
		fn);
	    // dustbin.txt/Block1226x03
	}}
	
	{{
	    // aggregating
	    if (m_aggOpt.size()){
		internal::vectorAggregateForward fn;

	    
		fn.dimInput2      = m_targetDim;      // 
		fn.dimInput2Start = m_targetDimStart; //

		fn.dim1Band       = m_targetDimEnd - m_targetDimStart; // dimension for 1 band
		fn.dimOutput      = this->size();
		fn.dimOutputStart = (this->precedingLayer().size() +
				     this->m_lookBack.size() * (m_targetDimEnd - m_targetDimStart));
		
		fn.input2         = helpers::getRawPointer(m_targetLayer->secondOutputs(true));
		fn.output         = helpers::getRawPointer(this->outputs());
		fn.bandNum        = this->m_aggOpt.size();
		
		fn.boundaryInfo   = helpers::getRawPointer(this->m_boundaryInfo);
		fn.startTime      = 0;
		fn.endTime        = this->curMaxSeqLength();
		
		fn.aggBuffer      = NULL;
		fn.crossBoundary  = m_crossBoundary;

		int n = (m_targetDimEnd - m_targetDimStart) * m_aggOpt.size();
		thrust::for_each(
			thrust::make_zip_iterator(
				thrust::make_tuple(this->outputs().begin(),
						   thrust::counting_iterator<int>(0))),
			thrust::make_zip_iterator(
				thrust::make_tuple(this->outputs().begin()+n,
						   thrust::counting_iterator<int>(0)+n)),
		fn);
	    }
	}}
	
	/*
	Cpu::real_vector tmp = this->outputs();
	for (int i = 0; i<this->curMaxSeqLength(); i++){
	    printf("%3d\t", i);
	    for (int j = 0; j < this->size(); j++){
		if (j<2)
		    printf("%0.2f\t", tmp[i*this->size() + j]);
		else
		    if (tmp[i*this->size() + j] * tmp[i*this->size() + j] > 0.00001)
			printf("One-hot: %3d", j);
	    }
	    printf("\n");
	}
	printf("\n");*/
    }

    // computeForwardPass
    // in synthesis stage, when the target must be predicted frame by frame
    template <typename TDevice>
    void FeedBackLayer<TDevice>::computeForwardPass(const int timeStep)
    {
	if (m_targetLayer == NULL){
	    throw std::runtime_error("Target layer is not linked");
	}	
	
	int effTimeStepS = timeStep     * this->parallelSequences();
	int effTimeStepE = (timeStep+1) * this->parallelSequences();
	int dimension    = 0;
	thrust::fill(this->outputs().begin() + effTimeStepS * this->size(), 
		     this->outputs().begin() + effTimeStepE * this->size(), 0.0);
	
	{{
	    // The dimension of the concatenated feature (if no softmax exists)
	    int previousSize  = this->precedingLayer().size();
	    
	    // Concatenate the feature vector 
	    // (by treating the 1 dimensional softmax Index as a normal feature)
	    internal::vectorFillForward fn;
	    
	    fn.dimInput1      = previousSize;
	    fn.dimInput2      = m_targetDim;
	    
	    fn.dimOutput      = this->size();
	    fn.parallel       = this->parallelSequences();
	    fn.dimInput2Start = m_targetDimStart;


	    fn.input1         = helpers::getRawPointer(this->precedingLayer().outputs());
	    fn.input2         = helpers::getRawPointer(m_targetLayer->secondOutputs(false));
	    fn.output         = helpers::getRawPointer(this->outputs());

	    fn.dim1Step       = m_targetDimEnd - m_targetDimStart; // dimension for 1 step
	    fn.lookBack       = helpers::getRawPointer(this->m_lookBack);

	    fn.lookBackStepNM = this->m_lookBack.size();
	    fn.crossBoundary  = m_crossBoundary;
	    thrust::for_each(
	       thrust::make_zip_iterator(
		 thrust::make_tuple(
			this->outputs().begin()+ effTimeStepS * this->size(),
			thrust::counting_iterator<int>(0)+ effTimeStepS * this->size())),
	       thrust::make_zip_iterator(
		 thrust::make_tuple(
			this->outputs().begin()+ effTimeStepE * this->size(),
			thrust::counting_iterator<int>(0)+ effTimeStepE * this->size())),
			fn);
	    // dustbin.txt/Block1226x04
	    
	}}

	{{
	    // aggregating
	    if (m_aggOptSyn==1 && m_aggOpt.size()){
		// strategy one:
		//   use the same aggration algorithm as in the training stage
		if (timeStep == 0)
		    thrust::fill(this->m_aggBuffer.begin(), this->m_aggBuffer.end(), 0.0);
		
		internal::vectorAggregateForwardInfer fn;
		fn.dimInput2      = m_targetDim;      // 
		fn.dimInput2Start = m_targetDimStart; //

		fn.dim1Band       = m_targetDimEnd - m_targetDimStart; // dimension for 1 band
		fn.dimOutput      = this->size();
		fn.dimOutputStart = (this->precedingLayer().size() +
				     this->m_lookBack.size() * (m_targetDimEnd - m_targetDimStart));
		fn.bandNum        = this->m_aggOpt.size();
				
		fn.input2         = helpers::getRawPointer(m_targetLayer->secondOutputs(false));
		fn.output         = helpers::getRawPointer(this->outputs());

		fn.boundaryInfo   = helpers::getRawPointer(this->m_boundaryInfo);
		fn.startTime      = timeStep;
		fn.endTime        = timeStep + 1;
		fn.aggBuffer      = helpers::getRawPointer(this->m_aggBuffer);;

		fn.crossBoundary  = m_crossBoundary;		
		dimension         = (m_targetDimEnd - m_targetDimStart) * m_aggOpt.size();

		
		thrust::for_each(
			thrust::make_zip_iterator(
				thrust::make_tuple(this->outputs().begin(),
						   thrust::counting_iterator<int>(0))),
			thrust::make_zip_iterator(
				thrust::make_tuple(this->outputs().begin()+dimension,
						   thrust::counting_iterator<int>(0)+dimension)),
						   fn);
	    }else if (m_aggOptSyn == 2){
		
		int previousSize  = this->precedingLayer().size();
		
		internal::vectorFillForward fn;
		
		fn.dimInput1      = previousSize;
		fn.dimInput2      = m_targetDim;
	    
		fn.dimOutput      = this->size();
		fn.parallel       = this->parallelSequences();
		fn.dimInput2Start = m_targetDimStart;
		

		fn.input1         = helpers::getRawPointer(this->precedingLayer().outputs());
		fn.input2         = helpers::getRawPointer(m_targetLayer->secondOutputs(false));
		fn.output         = helpers::getRawPointer(this->outputs());

		fn.dim1Step       = m_targetDimEnd - m_targetDimStart; // dimension for 1 step
		fn.crossBoundary  = m_crossBoundary;
		Cpu::int_vector  tmp(2,1);
		int_vector       tmpGPU = tmp;
		fn.lookBack       = helpers::getRawPointer(tmpGPU);

		fn.lookBackStepNM = m_aggOpt.size();
		
		thrust::for_each(
	         thrust::make_zip_iterator(
		  thrust::make_tuple(
			this->outputs().begin()+ effTimeStepS * this->size(),
			thrust::counting_iterator<int>(0)+ effTimeStepS * this->size())),
		 thrust::make_zip_iterator(
		  thrust::make_tuple(
			this->outputs().begin()+ effTimeStepE * this->size(),
			thrust::counting_iterator<int>(0)+ effTimeStepE * this->size())),
			fn);

	    }
	}}

    }

    // 
    template <typename TDevice>
    void FeedBackLayer<TDevice>::computeBackwardPass()
    {
	{{
	   // Copy the gradient for the preceding layer
	   internal::vectorFillBackward fn;
	   fn.dimInput1      = this->precedingLayer().size();
	   fn.dimOutput      = this->size();
	   fn.outputError    = helpers::getRawPointer(this->outputErrors());

	   int n = (this->curMaxSeqLength() * this->parallelSequences() *
		    this->precedingLayer().size());
	   
	   thrust::transform(thrust::counting_iterator<int>(0),
			     thrust::counting_iterator<int>(0)+n,
			     this->precedingLayer().outputErrors().begin(),
			     fn);	   
	}}
    }
    
    template class FeedBackLayer<Cpu>;
    template class FeedBackLayer<Gpu>;
    
}
